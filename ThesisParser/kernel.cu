#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <cstring> 
#include <stdlib.h>
#include <stdio.h>
#include <io.h>
#include <direct.h>
#include <string>
#include <windows.h>
#include "gloveparser.cuh"
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
#define GetCurrentDir _getcwd

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
const int glove_vector_count = 1193514;

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

int main()
{
	char filename[50], dim[5], currentdir[_MAX_PATH];
	int dimensions = 0;


	if (GetCurrentDir(currentdir, sizeof(currentdir)) != NULL) {
		printf("Current working directory: %s\n", currentdir);
	}
	else {
		perror("getcwd() error");
		return 1;
	}

	//Get the file name for the data set.
	printf("Enter filename: \n");
	fgets(filename, sizeof(filename), stdin);
	filename[strlen(filename) - 1] = '\0';
	char fullpath[_MAX_PATH];
	strcpy_s(fullpath, currentdir);
	strcat(fullpath, "\\datasets\\");
	strcat(fullpath, filename);
	printf("Full path is: %s\n", fullpath);

	//Get dimensions.
	printf("Enter number of dimensions: \n");
	fgets(dim, sizeof(dim), stdin);
	dimensions = atoi(dim);
	printf("Registered %d dimensions \n", dimensions);

	float* matrix = parseFile(fullpath, dimensions);

	for (int i = 0; i < 100; i++) {
		printf("%f", matrix[i]);
	}

	free(matrix);
	
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <1, size >> > (dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}
