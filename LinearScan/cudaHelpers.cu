#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <stdio.h>
#include "cudaHelpers.cuh"

void waitForKernel() {
	CUDA_CHECK_RETURN(hipGetLastError());
	CUDA_CHECK_RETURN(hipDeviceSynchronize());
}

void resetDevice() {
	CUDA_CHECK_RETURN(hipDeviceReset());
}

void setDevice(int device) {
	CUDA_CHECK_RETURN(hipSetDevice(device));
}