#include "hip/hip_runtime.h"
#include "point.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "point.h"
#include<iostream>
#include "pointExtensions.cuh"
#include <time.h>
#include <math.h>
#include "constants.cuh"
#include "sortParameters.h"
#include "sortingFramework.cuh"
#include "launchHelper.cuh"
#include "processingUtils.cuh"
#include "distanceFunctions.cuh"
#include "sketchedDistanceScanners.cuh"
#include "cudaHelpers.cuh"
#include "resultDTO.h"

__global__
void knn(float* queryPoints, float* dataPoints, int nQueries, int nData, int dimensions, int k, Point* result, int func) {

	Point threadQueue[THREAD_QUEUE_SIZE];
	int lane = threadIdx.x % WARPSIZE;
	Parameters params;
	params.lane = lane;
	int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / WARPSIZE;
	int resultIdx = warpId * k;
	int queryId = warpId * dimensions;
	if (warpId >= nQueries) return;
	float maxKDistance = (float)INT_MAX;
	int warpQueueSize = k / WARPSIZE;
	int candidateSetSize = THREAD_QUEUE_SIZE - warpQueueSize;
	int localMaxKDistanceIdx = THREAD_QUEUE_SIZE - warpQueueSize;
	Point swapPoint;
	int queuePosition = 0;

	//Fill thread queue with defaults
	for (int i = 0; i < THREAD_QUEUE_SIZE; i++) {
		threadQueue[i] = createPoint(-1, maxKDistance);
	}

	float magnitude_query = 0;


	for (int j = 0; j < dimensions; j++) {
		magnitude_query += queryPoints[queryId + j] * queryPoints[queryId + j];
	}

	magnitude_query = sqrt(magnitude_query);

	//Iterate over data; 
	for (int i = lane; i < nData; i += WARPSIZE) {
		float distance = 0.0;

		distance = runDistanceFunction(func, &dataPoints[i*dimensions], &queryPoints[queryId], dimensions, magnitude_query);

		Point currentPoint = createPoint(i, distance);

		if (WITH_TQ_OR_BUFFER) {
			//run TQ
			for (int j = candidateSetSize - 1; j >= 0; j--) { // simple sorting.
				if (currentPoint.distance < threadQueue[j].distance) {
					swapPoint = threadQueue[j];
					threadQueue[j] = currentPoint;
					currentPoint = swapPoint;
				}
			}


			//Verify that head of thread queue is not smaller than biggest k distance.
			if (__ballot_sync(FULL_MASK, threadQueue[0].distance < maxKDistance) && __activemask() == FULL_MASK) {
				startSort(threadQueue, swapPoint, params);
				maxKDistance = broadCastMaxK(threadQueue[candidateSetSize].distance);
			}
		}
		else {
			//run buffer
			if (currentPoint.distance < maxKDistance || same(currentPoint, maxKDistance)) {
				threadQueue[queuePosition++] = currentPoint;
			}



			if (__ballot_sync(FULL_MASK, queuePosition >= candidateSetSize) && __activemask() == FULL_MASK) {
				startSort(threadQueue, swapPoint, params);
				maxKDistance = broadCastMaxK(threadQueue[candidateSetSize].distance);
				//printQueue(threadQueue);
				queuePosition = 0;
			}
		}

	}

	startSort(threadQueue, swapPoint, params);

	//Copy result from warp queues to result array in reverse order. 
	int kIdx = (WARPSIZE - lane) - 1;
	int warpQueueIdx = THREAD_QUEUE_SIZE - 1;

	for (int i = kIdx; i < k; i += WARPSIZE)
	{
		result[resultIdx + i] = threadQueue[warpQueueIdx--];
	}

}

__global__
void normalizeData(float* queryPoints, float* dataPoints, int nQueries, int nData, int dimensions) {
	transformToUnitVectors(queryPoints, nQueries, dimensions);
	transformToUnitVectors(dataPoints, nData, dimensions);
}

__global__
void preprocess(float* queryPoints, float* dataPoints, int nQueries, int nData, int dimensions, int* minValues)
{
	transformData(dataPoints, queryPoints, nData, nQueries, dimensions, minValues);
}

__global__
void runScan(float* queryPoints, float* dataPoints, int nQueries, int nData, int dimensions, int k, Point* result, int func) {
	int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / WARPSIZE;
	int queryIndex = warpId * dimensions;
	if (warpId < nQueries) {
		scanHammingDistance(dataPoints, &queryPoints[queryIndex], dimensions, nullptr,nullptr, dimensions, nData, nQueries, k, func, 2, result);
	}
}

Result runMemOptimizedLinearScan(int k, int d, int N_query, int N_data, float* data, float* queries, int distanceFunc) {
	setDevice();
	int numberOfThreads = calculateThreadsLocal(N_query);
	int numberOfBlocks = calculateBlocksLocal(N_query);
	if (THREAD_QUEUE_SIZE <= 8 || THREAD_QUEUE_SIZE > 64) {
		numberOfThreads /= 2;
		numberOfBlocks *= 2;
	}
	int resultSize = N_query * k;
	Point *resultArray = (Point*)malloc(resultSize * sizeof(Point));
	Result res;
	res.setupResult(N_query, k);
	// queries
	float* dev_query_points = mallocArray(queries, N_query * d, true);
	// data
	float* dev_data_points = mallocArray(data, N_data * d, true);

	// result
	Point* dev_result = mallocArray(resultArray, resultSize);

	if (distanceFunc == 2) {
		printf("Starting preprocess \n");
		int* minValues = (int*)malloc(d * sizeof(int));
		for (int i = 0; i < d; i++) {
			minValues[i] = 0;
		}
		int* dev_minValues = mallocArray<int>(minValues, d, true);
		preprocess << <1, numberOfThreads >> > (dev_query_points, dev_data_points, N_query, N_data, d, dev_minValues);
		waitForKernel();

		normalizeData << < numberOfBlocks, numberOfThreads >> > (dev_query_points, dev_data_points, N_query, N_data, d);
		waitForKernel();

		printf("Done preprocessing \n");
	}

	printf("Launching KNN \n");
	size_t free_byte;
	size_t total_byte;
	hipMemGetInfo(&free_byte, &total_byte);
	double free_byte_double = (double)free_byte; 
	double totals_byte_double = (double)total_byte;
	double used_bytes = totals_byte_double - free_byte_double; 
	printf("Free bytes: %f, total_bytes: %f, used bytes %f \n", ((free_byte_double / 1024) / 1024), ((totals_byte_double / 1024) / 1024), ((used_bytes/1024)/1024));
	clock_t before = clock();
	knn << <numberOfBlocks, numberOfThreads>> > (dev_query_points, dev_data_points, N_query, N_data, d, k, dev_result, distanceFunc);
	waitForKernel();

	clock_t time_lapsed = clock() - before;
	printf("Time calculate on the GPU: %d \n", (time_lapsed * 1000 / CLOCKS_PER_SEC));
	res.scanTime = (time_lapsed * 1000 / CLOCKS_PER_SEC);
	copyArrayToHost(resultArray, dev_result, resultSize);
	res.copyResultPoints(resultArray, N_query, k);

	//Free memory... 
	freeDeviceArray(dev_query_points);
	freeDeviceArray(dev_data_points);
	freeDeviceArray(dev_result);
	free(resultArray);
	resetDevice();

	return res;
}