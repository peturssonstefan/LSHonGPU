#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"point.h"
#include"simHash.cuh"
#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <bitset>
#include <math.h>
#include <time.h>
#include "constants.cuh"
#include "hammingDistanceScanner.cuh"
#include "launchHelper.cuh"
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "processingUtils.cuh"
#include "cudaHelpers.cuh"
#include "statistics.cuh"
#include "statisticsCpu.h"
#include <map>

#define DISTANCE_FUNCTION 2

__global__
void transformVectors(float* data, float* queries, int N_data, int N_queries, int dimensions, int* m_bounds) {
	transformData(data, queries, N_data, N_queries, dimensions, m_bounds);
}

__global__
void normalizeVectors(float* data, float* queries, int N_data, int N_queries, int dimensions) {
	transformToUnitVectors(queries, N_queries, dimensions);
	transformToUnitVectors(data, N_data, dimensions);
}

__global__
void preprocess(float* data, float* queries, int N_data, int N_queries, int dimensions,int* m_bounds, int* m_indexMapSize) {
	int threadId = blockIdx.x * blockDim.x + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;
	
	// Find max
	for (int i = threadId; i < N_data; i += totalThreads) {
		for (int dim = 0; dim < dimensions; dim++) {
			atomicMax(&m_bounds[dim], ceil(data[i * dimensions + dim]));
		}
	}

	for (int i = threadId; i < N_queries; i += totalThreads) {
		for (int dim = 0; dim < dimensions; dim++) {
			atomicMax(&m_bounds[dim], ceil(queries[i * dimensions + dim]));
		}
	}

	__syncthreads();

	if (threadId == 0) {
		m_indexMapSize[0] = 0; 
		for (int i = 0; i < dimensions; i++) {
			m_indexMapSize[0] += m_bounds[i];
		}
	}

	__syncthreads();
}

__global__
void setupMapIndex(int* m_bounds, int* indexToComponentMap, int dimensions, int indexMapSize) {
	if (threadIdx.x == 0) { //TODO... We all know there is a smarter way to do this...
		int currentBound = 0;
		for (int i = 0; i < dimensions; i++) {
			int bound = currentBound + m_bounds[i];
			for (int j = currentBound; j < bound; j++) {
				if (j >= indexMapSize) {
					printf("j = %d and bounds[%d] = %d", j, i, m_bounds[i]);
				}
				else {
					indexToComponentMap[j] = i;
				}
			}

			m_bounds[i] = currentBound;
			currentBound = bound;
		}
	}
}

__inline__ __device__
float uniformRandom(hiprandState* state) {
	float val = hiprand_uniform(state); 
	return val; 
}

__inline__ __device__ 
bool isGreen(int* m_indexMap, int* m_bounds, float* data, float r, int i, int d) {
	int rIdx = r;
	int componentIdx = m_indexMap[rIdx];
	int m_bounds_val = m_bounds[componentIdx];
	float pointDI = data[i*d + componentIdx];
	return r <= m_bounds_val + pointDI; 

}


__global__
void sketchDataOneBit(float* data, int N_data, int dimensions, int sketchDim, int* m_indexMap, int* m_bounds, int M, int* seeds, bool* randomBitMap, unsigned char* sketchedData) {
	int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for (int i = threadId; i < N_data; i += totalThreads) {
		for (int hashIdx = 0; hashIdx < sketchDim; hashIdx++) {

			for (int bitIndex = 0; bitIndex < 8; bitIndex++) {
				int seed = seeds[hashIdx * 8 + bitIndex];
				hiprandState s;
				hiprand_init(seed, 0, 10000, &s);
				bool red = true;
				int counter = 0;
				float r = 0;
				while (red) {
					float random = uniformRandom(&s);
					r = M * random;
					red = !isGreen(m_indexMap, m_bounds, data, r, i, dimensions);
					if (red) {
						char val = sketchedData[i * sketchDim + hashIdx];
						counter++;
					}
				}
				int bit = randomBitMap[counter];
				sketchedData[i * sketchDim + hashIdx] |= bit << bitIndex;
			}


		}
	}

	//if (threadId == 0) {
	//	for (int i = 0; i < sketchDim * N_data; i++) {
	//		for (int bitIndex = 7; bitIndex >= 0; bitIndex--)
	//			printf("%d", (sketchedData[i] >> bitIndex) & 1);
	//			//printf("%d \n", sketchedData[i]);
	//		printf("\n");
	//	}
	//}

}

__global__
void sketchData(float* data, int N_data, int dimensions, int sketchDim, int* m_indexMap, int* m_bounds, int M, int* seeds, unsigned char* sketchedData) {
	int threadId = (blockIdx.x * blockDim.x) + threadIdx.x;
	int totalThreads = blockDim.x * gridDim.x;

	for (int i = threadId; i < N_data; i += totalThreads) {
		for (int hashIdx = 0; hashIdx < sketchDim; hashIdx++) {
			int seed = seeds[hashIdx];
			sketchedData[i * sketchDim + hashIdx] = 0;
			hiprandState s;
			hiprand_init(seed, 0, 10000, &s);
			bool red = true;
			while (red) {
				float random = uniformRandom(&s);
				float r = M * random;
				red = !isGreen(m_indexMap, m_bounds, data, r, i, dimensions);
				if (red) {
					sketchedData[i * sketchDim + hashIdx]++;
				}
			}
		}
	}
}


__global__
void scan(float* originalData, float* originalQueries, int dimensions, unsigned char * data, unsigned char * queries, int sketchDim, int N_data, int N_query, int k, Point* result) {
	int warpId = (blockIdx.x * blockDim.x + threadIdx.x) / WARPSIZE;
	int queryIndex = warpId * dimensions;
	if (queryIndex < dimensions * N_query) {
		scanHammingDistance(originalData, &originalQueries[queryIndex], dimensions, data, queries, sketchDim, N_data, N_query, k, DISTANCE_FUNCTION,result);
	}
}

__global__ 
void bucketDistributionKernel(unsigned char* hashes, int hashesSize, int* res) {
	bucketDistribution(hashes, hashesSize, res);

}

bool* generateRandomVectors(int N, bool randomSeed = false) {

	// same seed 
	static bool* vectors = (bool*)malloc(N * sizeof(bool));
	std::default_random_engine generator;
	// different seeds
	std::random_device rd;  // obtain a random number from hardware
	std::mt19937 eng(rd()); // seed the generator

	std::uniform_int_distribution<int> distribution(0, 1); // Standard normal distribution.

	for (int i = 0; i < N; ++i)
	{
		vectors[i] = distribution(randomSeed ? eng : generator);
		std::cout << vectors[i] << ",";
	}
	std::cout << std::endl; 
	return vectors;
}


Point* runWeightedMinHashLinearScan(int k, int d, int sketchedDim, int N_query, int N_data, float* data, float* queries, int implementation) {
	int numberOfThreads = calculateThreadsLocal(N_query);
	int numberOfBlocks = calculateBlocksLocal(N_query);
	
	int dataSize = d * N_data;
	int querySize = d * N_query;
	int resultSize = k * N_query;
	int charSize = 255; 
	int m_indexMapSize = 0;

	//Setup data array.
	float* dev_data = mallocArray(data, dataSize, true);
	
	//Setup query array.
	float* dev_queries = mallocArray(queries, querySize, true);
	
	bool runOneBitMinHash = implementation != 4; 

	//Seeds
	int seedArrSize = runOneBitMinHash ? sketchedDim * 8: sketchedDim;
	int* seedArr = (int*)malloc(seedArrSize * sizeof(int));

	for (int i = 0; i < seedArrSize; i++) {
		seedArr[i] = i * 1234 + 92138;
	}

	int* dev_seedArr = mallocArray(seedArr, seedArrSize, true);
	bool* randomBitMap = generateRandomVectors(charSize); 
	bool* dev_randomBitMap = mallocArray(randomBitMap, charSize, true); 
	//Sketch arrays
	int sketchedDataSize = N_data * sketchedDim;
	unsigned char* sketchedData = (unsigned char*)malloc(sketchedDataSize * sizeof(unsigned char));
	unsigned char* dev_sketchedData = mallocArray(sketchedData, sketchedDataSize);


	int sketchedQueriesSize = N_query * sketchedDim;
	unsigned char* sketchedQueries = (unsigned char*)malloc(sketchedQueriesSize * sizeof(unsigned char));
	unsigned char* dev_sketchedQueries = mallocArray(sketchedQueries, sketchedQueriesSize);


	int* m_bounds = (int*)malloc(d * sizeof(int));
	int* dev_m_bounds = mallocArray(m_bounds, d);

	int* m_indexMapSizeArr = (int*)malloc(sizeof(int));
	int* dev_m_IndexMapSizeArr = mallocArray(m_indexMapSizeArr, 1);

	// Transform data
	clock_t before = clock();

	transformVectors << <1, numberOfThreads >> > (dev_data, dev_queries, N_data, N_query, d, dev_m_bounds);
	waitForKernel();

	normalizeVectors << <numberOfBlocks, numberOfThreads >> > (dev_data, dev_queries, N_data, N_query, d);
	waitForKernel();

	preprocess << <1, numberOfThreads >> > (dev_data, dev_queries, N_data, N_query, d, dev_m_bounds, dev_m_IndexMapSizeArr);
	waitForKernel();
	clock_t time_lapsed = clock() - before;
	printf("Time to preprocess: %d \n", (time_lapsed * 1000 / CLOCKS_PER_SEC));

	copyArrayToHost(m_indexMapSizeArr, dev_m_IndexMapSizeArr, 1);
	m_indexMapSize = m_indexMapSizeArr[0];
	printf("Index map size: %d \n", m_indexMapSize);

	// Build maps
	int* m_IndexMap = (int*)malloc(m_indexMapSize * sizeof(int));
	int* dev_m_indexMap = mallocArray(m_IndexMap, m_indexMapSize);

	before = clock();
	setupMapIndex << <1, 1 >> > (dev_m_bounds, dev_m_indexMap, d, m_indexMapSize); //
	waitForKernel();
	time_lapsed = clock() - before;
	printf("Time to setup map: %d \n", (time_lapsed * 1000 / CLOCKS_PER_SEC));

	copyArrayToHost(m_IndexMap, dev_m_indexMap, m_indexMapSize);
	copyArrayToHost(m_bounds, dev_m_bounds, d);

	for (int i = 0; i < d; i++) {
		printf("%d ", m_bounds[i]);
	}
	printf("\n");
	for (int i = 0; i < m_indexMapSize; i++) {
		printf("%d ", m_IndexMap[i]);
	}
	printf("\n");

	printf("Starting sketch data \n");
	before = clock();

	if (runOneBitMinHash) {
		sketchDataOneBit << <numberOfBlocks, numberOfThreads >> > (dev_queries, N_query, d, sketchedDim, dev_m_indexMap, dev_m_bounds, m_indexMapSize, dev_seedArr, dev_randomBitMap,dev_sketchedQueries);
		waitForKernel();

		sketchDataOneBit << <numberOfBlocks, numberOfThreads >> > (dev_data, N_data, d, sketchedDim, dev_m_indexMap, dev_m_bounds, m_indexMapSize, dev_seedArr, dev_randomBitMap,dev_sketchedData);
		waitForKernel();
	}
	else {
		sketchData << <numberOfBlocks, numberOfThreads >> > (dev_queries, N_query, d, sketchedDim, dev_m_indexMap, dev_m_bounds, m_indexMapSize, dev_seedArr, dev_sketchedQueries);
		waitForKernel();

		sketchData << <numberOfBlocks, numberOfThreads >> > (dev_data, N_data, d, sketchedDim, dev_m_indexMap, dev_m_bounds, m_indexMapSize, dev_seedArr, dev_sketchedData);
		waitForKernel();
	}
	
	time_lapsed = clock() - before;
	printf("Time to hash on the GPU: %d \n", (time_lapsed * 1000 / CLOCKS_PER_SEC));
	
	printf("Done sketching \nStarting scan \n");
	int bucket_results_size = 255; 
	int* bucket_results = (int*)malloc(bucket_results_size * sizeof(int));
	int* bucket_results_dev = mallocArray(bucket_results, bucket_results_size);
	bucketDistributionKernel << <1, numberOfThreads >> > (dev_sketchedData, sketchedDataSize, bucket_results_dev);
	waitForKernel(); 


	copyArrayToHost(sketchedData, dev_sketchedData, sketchedDataSize);
	copyArrayToHost(sketchedQueries, dev_sketchedQueries, sketchedQueriesSize);

	copyArrayToHost(bucket_results, bucket_results_dev, bucket_results_size); 
	for (int i = 0; i < bucket_results_size; i++) {
		if (bucket_results[i] != 0) {
			printf("[%d] = %d \n", i, bucket_results[i]);
		}
	}

	//std::map<std::string, int> m = bucketDistributionFullKey(sketchedData, sketchedDataSize, sketchedDim); 

	//for (std::map<std::string,int>::iterator it = m.begin(); it != m.end(); ++it) {
	//	std::cout << it->first << " " << it->second << std::endl; 
	//}

	// Do linear scan
	Point* results = (Point*)malloc(resultSize * sizeof(Point));
	Point* dev_results = mallocArray(results, resultSize);

	before = clock();
	scan << <numberOfBlocks, numberOfThreads >> > (dev_data, dev_queries, d, dev_sketchedData, dev_sketchedQueries, sketchedDim, N_data, N_query, k, dev_results);
	waitForKernel();
	time_lapsed = clock() - before;
	printf("Time for scanning: %d \n", (time_lapsed * 1000 / CLOCKS_PER_SEC));

	copyArrayToHost(results, dev_results, resultSize);

	printf("Done with scan \n");
	//Close
	freeDeviceArray(dev_data);
	freeDeviceArray(dev_queries);
	freeDeviceArray(dev_sketchedData);
	freeDeviceArray(dev_sketchedQueries);
	freeDeviceArray(dev_m_bounds);
	freeDeviceArray(dev_results);
	free(sketchedData);
	free(sketchedQueries);
	free(m_bounds);

	resetDevice();

	return results;
}