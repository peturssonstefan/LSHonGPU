#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<math.h>
#include<iostream>
#include "gloveparser.cuh"
#include <stdio.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "point.h"
#include <time.h>
#include "simpleLinearScan.cuh"
#include "optimizedLinearScan.cuh"
#include "simHash.cuh"
#include "resultWriter.h"
#include "memOptimizedLinearScan.cuh"
#include "launchHelper.cuh"
#include "validation.h"
#include "weightedMinHash.cuh"


char* implementations[5] = { "OptimizedLinearScan", "MemOptimizedLinearScan", "SimHashLinearScan", "WeightedMinHash", "OneBit - WeightedMinHash" };

int main(int argc, char **argv)
{
	//In arguments. 
	char* filepath_data = argv[1];
	char* filepath_queries = argv[2];
	char* filepath_truth = argv[3]; 
	int shouldRunValidation = atoi(argv[4]);
	int writeRes = atoi(argv[5]); //1 for yes, 0 for no.
	char* _k = argv[6];
	
	int implementation = atoi(argv[7]);
	int reportK = atoi(_k);
	int k = calculateK(reportK);
	int distanceFunc = atoi(argv[9]); 
	int N_data = 0;
	int N_query = 0;
	int d = 0;
	float *queries;
	float *data;
	clock_t before = clock();
	queries = parseFile(filepath_queries, N_query, d);
	data = parseFile(filepath_data, N_data, d);
	clock_t time_lapsed = clock() - before;
	printf("Time to read data files: %d \n", (time_lapsed * 1000 / CLOCKS_PER_SEC));
	printf("Done parsing files. \n");
	printf("N_Query = %d \n", N_query);
	printf("N_Data = %d \n", N_data);
	printf("k is set to: %d\n", k);
	printf("Write res is set to %s \n", writeRes ? "True" : "False");
	printf("Validation is set to %s \n", shouldRunValidation ? "True" : "False");
	printf("Truth file is %s \n", filepath_truth);


	printf("Implementation selected = %s\n", implementations[implementation-1]); 
	Point* res; 

	switch (implementation)
	{
	case 1:
		res = runOptimizedLinearScan(k, d, N_query, N_data, data, queries);
		break;
	case 2:
		res = runMemOptimizedLinearScan(k, d, N_query, N_data, data, queries, distanceFunc);
		break;
	case 3: 
		res = runSimHashLinearScan(k, d, atoi(argv[8]), N_query, N_data, data, queries);
		break;
	case 4:
	case 5:
		printf("Running min hash \n");
		res = runWeightedMinHashLinearScan(k, d, atoi(argv[8]), N_query, N_data, data, queries, implementation);
		break;
	default:
		printf("Invalid implementation selected. \n");
		//exit(-1);
		break; //?
	}

	if (shouldRunValidation) {
		printf("Running Validation: \n");
		runValidation(filepath_truth, res, N_query, k, reportK); 
	}

	if (writeRes) {
		printf("Writing results: \n");
		writeResult(res, k, N_query, reportK);
	}

	printf("Starting to free \n"); 
	free(queries);
	free(data);
	printf("Success. Program exiting. \n");
	free(res);	
	return 0;

}

