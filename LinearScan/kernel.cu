#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<math.h>
#include<iostream>
#include "gloveparser.cuh"
#include <stdio.h>
#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>

__global__
void add(int n, int d, int k, float *x, float *y, float *z) {

	float dotProduct; 
	int queryIndex = threadIdx.x;
	int index = queryIndex * d;
	int z_index = queryIndex * k; 
	for (int i = 0; i < n; i++) {
		float dotProduct = 0; 
		float magnitude_x = 0.0;
		float magnitude_y = 0.0;
		for (int j = 0; j < d; j++) {
			dotProduct += x[index + j] * y[d*i + j];
			magnitude_x += x[index + j] * x[index + j];
			magnitude_y += y[d*i + j] * y[d*i + j];
		}

		magnitude_x = sqrt(magnitude_x);
		magnitude_y = sqrt(magnitude_y);
		float angular_distance = -(dotProduct / (magnitude_x * magnitude_y));
		//z[z_index + i] = angular_distance; 
		float tmp_distance = 0; 
		for (int j = 0; (j < k && j < i); j++) { // simple sorting.
			if (z[z_index + j] > angular_distance) {
				tmp_distance = z[z_index + j]; 
				z[z_index + j] = angular_distance; 
				angular_distance = tmp_distance; 
			}
		}
	}
}


int main(int argc, char **argv)
{
	char* filepath_data = argv[1];
	char* filepath_queries = argv[2];
	char* _k = argv[3];
	int k = atoi(_k);

	int N_data = 0;
	int N_query = 0; 
	int d = 0;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}
	float *x;
	float *y;
	float *z;
	printf("Parsing files... \n");
	x = parseFile(filepath_queries, N_query, d); 
	y = parseFile(filepath_data, N_data, d);
	printf("Done parsing files. \n");
	printf("N_Query = %d \n", N_query);
	printf("N_Data = %d \n", N_data); 
	printf("k is set to: %d\n", k);
	z = (float*)malloc(k*N_query * sizeof(float));
	
	for (int i = 0; i < k * N_query; i++) {
		z[i] = 2.0f; //fill z array with default max value. 
	}

	float* dev_x = 0;
	float* dev_y = 0;
	float* dev_z = 0;
	hipMalloc((void**)&dev_x, N_query * d * sizeof(float));
	hipMalloc((void**)&dev_y, N_data * d * sizeof(float));
	hipMalloc((void**)&dev_z, k * N_query * sizeof(float));

	hipMemcpy(dev_x, x, N_query * d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, N_data * d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_z, z, k * N_query * sizeof(float), hipMemcpyHostToDevice);
	// initialize x and y arrays on the host

	add << <1, N_query>> > (N_data, d, k, dev_x, dev_y, dev_z);



	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return -1;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return -1;
	}

	cudaStatus = hipMemcpy(z, dev_z, k * N_query * sizeof(float), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cuda memcpy from device to host returned error code %d \n", cudaStatus);
		return -1;
	}

	for (int i = 0; i < k*N_query; i++) { 
		printf("z[%d] = %f\n", i, z[i]); 
	}

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	free(x);
	free(y);

	cudaStatus = hipDeviceReset();
	return 0;
}

