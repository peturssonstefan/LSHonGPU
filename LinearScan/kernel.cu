#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include<math.h>
#include<iostream>

#include <stdio.h>

__global__
void add(int n, int d, float *x, float *y, float *z) {
	 

	float dotProduct; 
	int queryIndex = threadIdx.x;
	int index = queryIndex * d;
	for (int i = 0; i < n; i++) {
		float dotProduct = 0; 
		float magnitude_x = 0.0;
		float magnitude_y = 0.0;
		for (int j = 0; j < d; j++) {
			dotProduct += x[queryIndex + j] * y[d*i + j];
			magnitude_x += x[queryIndex + j] * x[queryIndex + j]; 
			magnitude_y += y[d*i + j] * y[d*i + j];
		}

		magnitude_x = sqrt(magnitude_x);
		magnitude_y = sqrt(magnitude_y);
		z[queryIndex * n + i] = dotProduct / (magnitude_x * magnitude_y); 
	}
}


int main()
{
	const int N_data = 5;
	const int N_query = 5; 
	const int d = 5;
	hipError_t cudaStatus;
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return -1;
	}
	float *x;
	float *y;
	float *z;

	x = (float*)malloc(N_query * d * sizeof(float));
	y = (float*)malloc(N_data * d * sizeof(float));
	z = (float*)malloc(N_data*N_query * sizeof(float)); 

	for (int i = 0; i < N_query * d; i++) {
		x[i] = 1.0f;
	}

	for (int i = 0; i < N_data*d; i++) {
		y[i] = 2.0f;
	}

	float* dev_x = 0;
	float* dev_y = 0;
	float* dev_z = 0;
	hipMalloc((void**)&dev_x, N_query * d * sizeof(float));
	hipMalloc((void**)&dev_y, N_data * d * sizeof(float));
	hipMalloc((void**)&dev_z, N_data * N_query * sizeof(float));

	hipMemcpy(dev_x, x, N_query * d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, y, N_data * d * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_z, y, N_data * N_query * sizeof(float), hipMemcpyHostToDevice);
	// initialize x and y arrays on the host

	add << <1, N_query>> > (N_data, d, dev_x, dev_y, dev_z);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		return -1;
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return -1;
	}

	cudaStatus = hipMemcpy(z, dev_z, N_data * N_query * sizeof(float), hipMemcpyDeviceToHost);

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		return -1;
	}

	for (int i = 0; i < N_data*N_query; i++) {
		printf("z[%d] = %f \n", i, z[i]);
	}

	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	free(x);
	free(y);

	cudaStatus = hipDeviceReset();
	return 0;
}

